#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define TILE_DIM 4
#define FILTER_RADIUS 3
#define FILTER_SIZE (2 * FILTER_RADIUS + 1)
#define OUT_TILE_DIM (TILE_DIM - 2 * FILTER_RADIUS)
__constant__ float F[FILTER_SIZE * FILTER_SIZE];
__global__ void convolution_cached_tiled_2D_kernel(float* N, float* P, int width, int height) {
  __shared__ float N_s[TILE_DIM][TILE_DIM];
  int col = blockIdx.x * OUT_TILE_DIM + threadIdx.x;
  int row = blockIdx.y * OUT_TILE_DIM + threadIdx.y;
  if (row < height && col < width) {
    N_s[threadIdx.y][threadIdx.x] = N[row * width + col];
  } else {
    N_s[threadIdx.y][threadIdx.x] = 0.0f;
  }
  __syncthreads();
  col = blockIdx.x * OUT_TILE_DIM + threadIdx.x - FILTER_RADIUS;
  row = blockIdx.y * OUT_TILE_DIM + threadIdx.y - FILTER_RADIUS;
  if (col >= 0 && col < width && row >= 0 && row < height) {
    float Pvalue = 0.0f;
    for (int fRow = 0; fRow < FILTER_SIZE; fRow++) {
      for (int fCol = 0; fCol < FILTER_SIZE; fCol++) {
        int row_offset = threadIdx.y - FILTER_RADIUS + fRow;
        int col_offset = threadIdx.x - FILTER_RADIUS + fCol;
        if (row_offset >= 0 && row_offset < TILE_DIM && col_offset >= 0 && col_offset < TILE_DIM) {
          int global_row = row - FILTER_RADIUS + fRow;
          int global_col = col - FILTER_RADIUS + fCol;
          if (global_row >= 0 && global_row < height && global_col >= 0 && global_col < width) {
            Pvalue += F[fRow * FILTER_SIZE + fCol] * N_s[row_offset][col_offset];
          }
        }
      }
    }
    P[row * width + col] = Pvalue;
  }
}
void print_matrix(float* matrix, int width, int height, const char* name) {
  printf("%s Matrix:\n", name);
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      printf("%.2f ", matrix[i * width + j]);
    }
    printf("\n");
  }
  printf("\n");
}
int main() {
  hipEvent_t total_start, total_stop, kernel_start, kernel_stop;
  hipEventCreate(&total_start);
  hipEventCreate(&total_stop);
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);
  float total_milliseconds = 0, kernel_milliseconds = 0;
  hipEventRecord(total_start);
  const char* input_path = "IMG2.jpg";
  int width, height, channels;
  unsigned char* image = stbi_load(input_path, &width, &height, &channels, 0);
  if (!image) {
    printf("Error loading image from %s\n", input_path);
    return 1;
  }
  float* h_input = (float*)malloc(width * height * sizeof(float));
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      h_input[i * width + j] = image[i * width * channels + j * channels] / 255.0f;
    }
  }
  print_matrix(h_input, width, height, "Original Image");
  float* h_output = (float*)malloc(width * height * sizeof(float));
  float h_filter[FILTER_SIZE * FILTER_SIZE] = {
    1.0f / 256, 4.0f / 256, 6.0f / 256, 4.0f / 256, 1.0f / 256, 0.0f / 256, 0.0f / 256,
    4.0f / 256, 16.0f / 256, 24.0f / 256, 16.0f / 256, 4.0f / 256, 0.0f / 256, 0.0f / 256,
    6.0f / 256, 24.0f / 256, 36.0f / 256, 24.0f / 256, 6.0f / 256, 0.0f / 256, 0.0f / 256,
    4.0f / 256, 16.0f / 256, 24.0f / 256, 16.0f / 256, 4.0f / 256, 0.0f / 256, 0.0f / 256,
    1.0f / 256, 4.0f / 256, 6.0f / 256, 4.0f / 256, 1.0f / 256, 0.0f / 256, 0.0f / 256,
    0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256,
    0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256, 0.0f / 256
  };
  float* d_input, * d_output;
  hipMalloc(&d_input, width * height * sizeof(float));
  hipMalloc(&d_output, width * height * sizeof(float));
  size_t input_bytes = width * height * sizeof(float);
  size_t output_bytes = width * height * sizeof(float);
  size_t filter_bytes = FILTER_SIZE * FILTER_SIZE * sizeof(float);
  size_t host_to_device_bytes = input_bytes + filter_bytes;
  size_t device_to_host_bytes = output_bytes;
  size_t tiles_per_row = (width + OUT_TILE_DIM - 1) / OUT_TILE_DIM;
  size_t tiles_per_col = (height + OUT_TILE_DIM - 1) / OUT_TILE_DIM;
  size_t total_tiles = tiles_per_row * tiles_per_col;
  size_t kernel_reads_global = total_tiles * TILE_DIM * TILE_DIM * sizeof(float);
  size_t kernel_writes = width * height * sizeof(float);
  size_t kernel_total_bytes = kernel_reads_global + kernel_writes;
  hipMemcpy(d_input, h_input, input_bytes, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(F), h_filter, filter_bytes);
  dim3 blockDim(TILE_DIM, TILE_DIM);
  dim3 gridDim((width + OUT_TILE_DIM - 1) / OUT_TILE_DIM, (height + OUT_TILE_DIM - 1) / OUT_TILE_DIM);
  hipEventRecord(kernel_start);
  convolution_cached_tiled_2D_kernel << <gridDim, blockDim >> > (d_input, d_output, width, height);
  hipEventRecord(kernel_stop);
  hipEventSynchronize(kernel_stop);
  hipEventElapsedTime(&kernel_milliseconds, kernel_start, kernel_stop);
  hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);
  print_matrix(h_output, width, height, "Blurred Image");
  unsigned char* output_image = (unsigned char*)malloc(width * height);
  for (int i = 0; i < width * height; i++) {
    float val = h_output[i];
    if (val < 0.0f) val = 0.0f;
    if (val > 1.0f) val = 1.0f;
    output_image[i] = (unsigned char)(val * 255.0f);
  }
  const char* output_path = "C:\\Users\\N01538486\\Desktop\\blurred_output_tiled.png";
  if (!stbi_write_png(output_path, width, height, 1, output_image, width)) {
    printf("Error saving image to %s\n", output_path);
  } else {
    printf("Blurred image saved to %s\n", output_path);
  }
  hipEventRecord(total_stop);
  hipEventSynchronize(total_stop);
  hipEventElapsedTime(&total_milliseconds, total_start, total_stop);
  float total_seconds = total_milliseconds / 1000.0f;
  float kernel_seconds = kernel_milliseconds / 1000.0f;
  float host_to_device_bw = (host_to_device_bytes / (1024.0f * 1024.0f * 1024.0f)) / total_seconds;
  float device_to_host_bw = (device_to_host_bytes / (1024.0f * 1024.0f * 1024.0f)) / total_seconds;
  float kernel_bw = (kernel_total_bytes / (1024.0f * 1024.0f * 1024.0f)) / kernel_seconds;
  float total_bw = ((host_to_device_bytes + device_to_host_bytes + kernel_total_bytes) / (1024.0f * 1024.0f * 1024.0f)) / total_seconds;
  printf("\nMemory Bandwidth Measurements:\n");
  printf("Kernel Execution Time: %.3f ms\n", kernel_milliseconds);
  printf("Total Execution Time: %.3f ms\n", total_milliseconds);
  printf("Host to Device Bandwidth: %.3f GB/s\n", host_to_device_bw);
  printf("Device to Host Bandwidth: %.3f GB/s\n", device_to_host_bw);
  printf("Kernel Memory Bandwidth: %.3f GB/s\n", kernel_bw);
  printf("Total Effective Bandwidth: %.3f GB/s\n", total_bw);
  printf("Total Bytes Transferred: %.3f MB\n", (host_to_device_bytes + device_to_host_bytes + kernel_total_bytes) / (1024.0f * 1024.0f));
  hipEventDestroy(kernel_start);
  hipEventDestroy(kernel_stop);
  hipEventDestroy(total_start);
  hipEventDestroy(total_stop);
  stbi_image_free(image);
  free(h_input);
  free(h_output);
  free(output_image);
  hipFree(d_input);
  hipFree(d_output);
  return 0;
}
